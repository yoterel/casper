#include "hip/hip_runtime.h"
// #include <hip/hip_cooperative_groups.h>
#include <hip/hip_runtime_api.h>

hipTextureObject_t inTexObject;

// get pixel from 2D image, with clamping to border
__device__ float4 getPixel(int x, int y, hipTextureObject_t inTex) {
  float4 res = tex2D<float4>(inTex, x, y);
  return res;
}


__global__ void cudaProcess(float *g_odata, int imgw, int imgh,
                            int tilew, int r, float threshold, float highlight,
                            hipTextureObject_t inTex) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x * bw + tx;
    int y = blockIdx.y * bh + ty;
    g_odata[y * imgw + x] = getPixel(x, y, inTex);
}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes,
                                   hipArray *g_data_array,
                                   float *g_odata, int imgw, int imgh,
                                   int tilew, int radius, float threshold,
                                   float highlight) {
  struct hipChannelFormatDesc desc;
  checkCudaErrors(hipGetChannelDesc(&desc, g_data_array));

  hipResourceDesc texRes;
  memset(&texRes, 0, sizeof(hipResourceDesc));

  texRes.resType = hipResourceTypeArray;
  texRes.res.array.array = g_data_array;

  hipTextureDesc texDescr;
  memset(&texDescr, 0, sizeof(hipTextureDesc));

  texDescr.normalizedCoords = false;
  texDescr.filterMode = hipFilterModePoint;
  texDescr.addressMode[0] = hipAddressModeWrap;
  texDescr.readMode = hipReadModeElementType;

  checkCudaErrors(
      hipCreateTextureObject(&inTexObject, &texRes, &texDescr, NULL));

    cudaProcess<<<grid, block, sbytes>>>(g_odata, imgw, imgh,
                                         block.x + (2 * radius), radius, 0.8f,
                                         4.0f, inTexObject);

}
